#include "hip/hip_runtime.h"
CDF inline bool hasShadow(trailword in){return (bool)(in.var&SEEN_FLAG);};
CDF inline bool isDecision(trailword in){return (bool)(in.var&DECISION_FLAG);};
CDF inline varind getIndex(trailword in){return (varind)((in.var&VARINDEX_MASK)>>1);};
/*
CDF inline varind getIndex(trailword in){
	varind tmp;
	tmp = (varind)((in.var&VARINDEX_MASK)>>1);
	assert (tmp
	return tmp; };
	*/

CDF bool DBGVerifyTrail(SDD)
{
	for (int i=1; i<=trail_end; i++){
		for (int j=1; j<i; j++){
			if (getIndex(trail[stride*i+shard_num])==getIndex(trail[stride*j+shard_num])){
				printf("\n TRAIL (%i) %i %i = %i %i", trail_end, i, getIndex(trail[i]), j, getIndex(trail[j]));
				return true;
			}
		}
	}
	return false;
}


CDF inline int bindex(int b) { return b / WORD_SIZE; }
CDF inline int boffset(int b) { return b & (uint)(WORD_SIZE-1); }
CDF inline void SetBitStrided(SDD,var_word* data, int b) { 
	//atomicOr(&data[stride*bindex(b)+shard_num], 1 << (boffset(b))); 
	data[stride*bindex(b)+shard_num]|= (1 << (boffset(b))); 
}
CDF inline void ClearBitStrided(SDD, var_word* data, int b) { 
	//atomicAnd(&data[stride*bindex(b)+shard_num], ~(1 << (boffset(b))));
	data[stride*bindex(b)+shard_num]&= ~(1 << (boffset(b)));
}
CDF inline bool GetBitStrided(SDD, const var_word* data, int b) { 
	return (bool)(data[stride*bindex(b)+shard_num] & (1 << (boffset(b))));
}
CDF inline  void SetBitSimple(SDD, var_word* data, int b) { 
	//atomicOr(&data[bindex(b)], 1 << (boffset(b))); 
	data[bindex(b)]|= (1 << (boffset(b))); 
}
CDF inline  void ClearBitSimple(SDD, var_word* data, int b) { 
	//atomicAnd(&data[bindex(b)], ~(1 << (boffset(b))));
	data[bindex(b)] &= ~(1 << (boffset(b)));
}
CDF inline bool GetBitSimple(SDD, const var_word* data, int b) { 
	return (bool)(data[bindex(b)] & (1 << (boffset(b))));
}
CDF inline void SetVarset(SDD, int b){
	//assert(b>0);
	//assert(b<c.pl_size/2);
	SetBitStrided(SDA,  vars, b*2);
}
CDF inline void SetVarphase(SDD, int b){
	//assert(b>0);
	//assert(b<c.pl_size/2);
	SetBitStrided(SDA,  vars, b*2+1);
}
CDF inline void ClearVarset(SDD, int b){
	//assert(b>0);
	//assert(b<c.pl_size/2);
	ClearBitStrided(SDA,  vars, b*2);
}
CDF inline void ClearVarphase(SDD, int b){
	//assert(b>0);
	//assert(b<c.pl_size/2);
	ClearBitStrided(SDA,  vars, b*2+1);
}
CDF inline bool GetVarset(SDD,  int b){
	//assert(b>0);
	//assert(b<c.pl_size/2);
	return GetBitStrided(SDA,  vars, b*2);
}
CDF inline bool GetVarphase(SDD, int b)
{
	/*
	assert(b>0);
	assert(b<c.pl_size/2);
	*/
	return GetBitStrided(SDA,  vars, b*2+1);
}

CDF inline void ClearVar(SDD, int b)
{
	//assert(b>0);
	//assert(b<c.pl_size/2);
	//TODO: сделать одной командой!!
	ClearVarphase(SDA,  b);
	ClearVarset(SDA,  b);
	//data[bindex(b*2)] &= ~(3 << (boffset(b*2)));
}

CDF inline int VarToHVar(const varind var){return (var>>1)*(-1+2*(var&1));}
CDF inline varind HVarToVar(const int var)
{
	/*
	assert(var!=0);
	assert (((varind)((abs(var)<<1)|(var<0)))>0);
	*/
	return (varind)((abs(var)<<1)|(var>0));
}

CDF inline bool VarPositive(const varind var){return (bool)(var&1);}

CDF inline int LitToHVar(const Rcnf& c, litind ind){ return VarToHVar(c.lits[ind].y); }

CDF void DBGPrintClauseFromRing(
		const Rcnf& c, 
		litind pos, 
		litind startpos = NO_LITIND, 
		bool top = true)
{
	//assert(pos < c.lits_size);
	if (top){
		startpos = pos;
	}
	printf("t%i:%i ", threadIdx.x, LitToHVar(c, pos));
	if (c.lits[pos].x==startpos){
	//	printf("\n");
	}else{
		DBGPrintClauseFromRing(c, c.lits[pos].x, startpos, false);
	}
}
CDF inline bool isBCPQueueEmpty(SDD)
{
	return (BCP_queue_front>trail_end);
}


CDF bool searchTrail(SDD, const int var)
{
	for (int i=1; i<=trail_end; i++){
		if (getIndex(trail[stride*i+shard_num])==abs(var)){
			return true;
		}
	}
	return false;
}

/*
CDF bool verifytrail(SDD){
	for (int i=1; i<=trail_end; i++){
		for (int j=1; j<i; j++){
			if (getIndex(trail[stride*i+shard_num])==getIndex(trail[stride*j+shard_num])){
				printf("\n trail %i %i", i , j);
				return true;
			}
		}
	}
	return false;
}
*/

CDF bool DBGVerifyVars(SDD){
	for (int i=1; i<=trail_end; i++){
		if (!GetVarset(SDA, getIndex(trail[stride*i+shard_num]))){
			printf("\n No var  trail%i ind%i", i, getIndex(trail[stride*i+shard_num]));
			return true;
		}
	}
	return false;
}

CDF void RemoveWatches(SDD, const litind firstlit)
{
	litind current=0;
	current = firstlit;
	do{
		//ClearBitStrided(SDA, ws, current);
		current=c.lits[current].x;
		//printf("lit %i-%i", current, get_bit(ws,current));
	} while(current != firstlit);
}

CDF inline void BCPQueuePopFront(SDD){BCP_queue_front++;}

CDF inline void BCPQueueClear(SDD){BCP_queue_front=trail_end+1;}

/*
CDF int verifyClauseR(
		SDD,
		TR_ARGS_DEF,
		uint &inspects,
		const Rcnf& c,
		var_word* ws,
		const litind firstlit)
{
	litind current=0;
	current = firstlit;
	int wseen=0;
	//printf("\n");
	do{
		wseen+=GetBitSimple(sd, shard_num, &ws[(c.lits_size/WORD_SIZE+1)*shard_num], current);
		current=c.lits[current].x;
		//printf("lit %i-%i", current, get_bit(ws,current));
	} while(current != firstlit);
	//assert(wseen!=0);
	//assert(wseen!=1);
	//assert(wseen==2);
	return wseen;
}
*/

//TODO: microoptimize!
CDF void SetVar(SDD, const varind ind,
	       	const litind reason_lit = 0, const bool decision_var=false)
{
	//if (shard_num==0 && abs(VarToHVar(ind))==128) printf("\n set var 128 to %i reason %i", ind, reason_lit);
	//if (shard_num==0) printf("\n set var to %i reason %i", ind, reason_lit);
	assert(ind>0);
	assert((ind>>1)<c.pl_size/2);
	assert(!GetVarset(SDA, ind>>1));
	SetVarset(SDA, ind>>1);
	if (VarPositive(ind)){
		SetVarphase(SDA, ind>>1);
	}else{
		ClearVarphase(SDA, ind>>1);
	}
	trailword new_trail_element;
		
	if(decision_var){
		new_trail_element.var=ind |DECISION_FLAG;
		/*
		assert(getIndex(new_trail_element)<c.pl_size/2);
		assert(isDecision(new_trail_element));
		assert(getIndex(new_trail_element)<c.pl_size/2);
		assert(getIndex(new_trail_element)==ind>>1);
		*/
	}else{
		assert(reason_lit>0);
		new_trail_element.var=ind;
	}
	assert(reason_lit>=0);

	assert(getIndex(new_trail_element)<c.pl_size/2);
	assert(reason_lit<c.lits_size);
#ifdef BJ
	new_trail_element.reason=reason_lit;
#endif //BJ
	++trail_end;
	assert(trail_end<c.pl_size/2);
	trail[stride*trail_end+shard_num]=new_trail_element;
}

CDF void SetVarRemove(SDD, const varind ind)
{
	SetVar(SDA, ind);
	/*
	for (int i=c.pl[ind-1]; i<c.pl[ind]; i++){
		if (c.lits[i].x!=NO_LITIND && c.lits[i].y!=0){
			RemoveWatches(SDA, i);
		}
	}
	*/
}

CDF bool DBGPrintClause(SDD, const litind firstlit)
{
	printf("\n");
	litind current_ind = firstlit;

	do{
		assert(c.lits[current_ind].x!=current_ind);
		uintx current_lit = c.lits[current_ind];	
		printf(" %i.%i:(%i|%iw%i)", current_lit.x,
				VarToHVar(current_lit.y),
				GetVarset(SDA,
					current_lit.y>>1),
				GetVarphase(SDA,
					current_lit.y>>1),
				GetBitStrided(SDA, ws,
					current_ind)
				);
		current_ind=c.lits[current_ind].x;
	}while(current_ind!=firstlit);
	return true;
}

CDF bool clauseSolved(SDD, const litind firstlit)
{
	litind next = c.lits[firstlit].x;
	uintx next_lit = c.lits[next];
	while (true){
		litind current = next;
		uintx current_lit=next_lit;
		next = current_lit.x;
		next_lit = c.lits[next];
		uint var_ind = current_lit.y>>1;
		var_word var_container = vars[stride*bindex(var_ind*2)+shard_num];
		bool var_set= (bool)( var_container & (1 << (boffset(var_ind*2))));
		bool var_phase= (bool)(var_container & (1 << (boffset(var_ind*2+1))));
		bool lit_phase = (bool)(current_lit.y&1);
		bool lit_solving = ((var_phase == lit_phase) && var_set);
		if (lit_solving){
			return true;
		}
		if(current==firstlit){
			return false;
		}
	}
}

CDF bool clauseEmpty(SDD, const litind firstlit)
{
	litind next = c.lits[firstlit].x;
	uintx next_lit = c.lits[next];
	while (true){
		litind current = next;
		uintx current_lit=next_lit;
		next = current_lit.x;
		next_lit = c.lits[next];
		uint var_ind = current_lit.y>>1;
		var_word var_container = vars[stride*bindex(var_ind*2)+shard_num];
		bool var_set= (bool)( var_container & (1 << (boffset(var_ind*2))));
		bool var_phase= (bool)(var_container & (1 << (boffset(var_ind*2+1))));
		bool lit_phase = (bool)(current_lit.y&1);
		bool lit_solving = ((var_phase == lit_phase) && var_set);
		if(!var_set){
			return false;
		}
		if (lit_solving){
			return false;
		}
		if(current==firstlit){
			return true;
		}
	}
}

CDF bool clauseUnit(SDD, const litind firstlit)
{
	litind next = c.lits[firstlit].x;
	uintx next_lit = c.lits[next];
	int free_lit_counter=0;
	while (true){
		litind current = next;
		uintx current_lit=next_lit;
		next = current_lit.x;
		next_lit = c.lits[next];
		uint var_ind = current_lit.y>>1;
		var_word var_container = vars[stride*bindex(var_ind*2)+shard_num];
		bool var_set= (bool)( var_container & (1 << (boffset(var_ind*2))));
		bool var_phase= (bool)(var_container & (1 << (boffset(var_ind*2+1))));
		bool lit_phase = (bool)(current_lit.y&1);
		bool lit_solving = ((var_phase == lit_phase) && var_set);
		if (lit_solving){
			return false;
		}
		if (!var_set){
			free_lit_counter++;
		}
		if(current==firstlit){
			break;
		}
	}
	if(free_lit_counter==1){
		return true;
	}
	return false;
}

CDF int CheckClause(SDD, const litind firstlit)
{
	litind ws2_ind=0;
	litind ws1_new=0, ws2_new=0;
	varind ws1_var=0, ws2_var=0;
	assert(firstlit<c.lits_size);
	LIT_STATE ws1_state=NONFREE;
	LIT_STATE ws2_state=UNKNOWN;
	//uint2 current_lit=c.lits[firstlit];

	litind next = c.lits[firstlit].x;
	uintx next_lit = c.lits[next];

	//litind lastlit_ind = firstlit;
	int inspects_local=0;
	//prefetch через двойной буфер по вотчам и фазам только ухудшает
	//производительность
	
	//if (shard_num==3 && firstlit==19266) printf("\n lit.y %i", c.lits[firstlit].y);
	//условия выхода из цикла: кончились литералы или BCP уже точно не будет
	while (true){
		//litind current = current_lit.x;
		litind current = next;
		uintx current_lit=next_lit;
		if(current==firstlit){
			break;
		}
		next = current_lit.x;
		next_lit = c.lits[next];
		//assert(current<c.lits_size);
		inspects_local++;
		inspects.x++;
		//current_lit=c.lits[current];
		// TODO: оптимизировать работу со сдвигами
		uint var_ind = current_lit.y>>1;
		var_word var_container = vars[stride*bindex(var_ind*2)+shard_num];
		//var_word var_container = vars[warpIdx()][bindex(var_ind*2)];
		bool var_set= (bool)( var_container & (1 << (boffset(var_ind*2))));
		bool var_phase= (bool)(var_container & (1 << (boffset(var_ind*2+1))));
		//bool lit_phase = getLitPhase(c,current);
		bool lit_phase = (bool)(current_lit.y&1);
		bool lit_solving = ((var_phase == lit_phase) && var_set);
		bool lit_watched = false;
		// с включением этой оптимизации производительность падает на 10% (((((
		//if(ws2_state==UNKNOWN){
		lit_watched= GetBitStrided(SDA, ws,current);
		//}

		if (lit_watched){
			if (lit_solving){
				inspects.y+=inspects_local;
				return 0;
			}
			ws2_ind=current;
			ws2_var=current_lit.y;
			if (!lit_solving && var_set){
				ws2_state = NONFREE;
			}else{
				ws2_state = FREE;
			}
			continue;
		}

		/*
		if (lit_watched){
			if (lit_solving){
				if (inspects_local>1) inspects.y+=inspects_local;
				//ws1_new=lastlit_ind;
				//ws1_state=SOLVING;
				//break;
				return 0;
			}
			ws2_ind=current;
			ws2_var=current_lit.y;
			if (!lit_solving && var_set){
				ws2_state = NONFREE;
			}else{
				ws2_state = FREE;
			}
			continue;
		}
		*/
		//lastlit_ind = current;

		//литерал выполняет дизъюнкт
		if (lit_solving){
			// поменять первый вотч
		/*	
			if(ws2_state==NONFREE){
				ws2_new=lastlit_ind;
			}
		*/	
			ws1_new=current;
			ws1_var=current_lit.y;
			ws1_state=SOLVING;
			break;
		}
		//литерал несвободен
		if (var_set){
			continue;
		}
		//литерал свободен, второй вотч неизвестен или (известен и свободен)
		//поменять первый вотч
		if (ws1_new==0){
			ws1_new=current;
			ws1_var=current_lit.y;
			if (ws2_state!=NONFREE){
				break;
			}
			continue;
		}
		//литерал свободен, второй вотч известен и несвободен
		if (ws2_state==NONFREE){
			ws2_new=current;
			ws2_var=current_lit.y;
			break;
		}
		/*	
		printf("\n Thread:%i lit %i var %i ws %i vset %i vstate %i",  threadIdx.x, current,
				LitToHVar(c, current),
				GetBitSimple(SDA, shard_num, &ws[(c.lits_size/WORD_SIZE+1)*shard_num],current), var_set, var_phase);
		*/
	}
	//assert(!GetVarset(SDA, shard_num, (c.lits[ws2_ind].y)>>1));
	//assert(((c.lits[ws2_ind].y)>>1)!=((c.lits[firstlit].y)>>1));
	/*
	if (shard_num==3 && firstlit==19266){ printf("\n ws1_new %i ws2_new %i", ws1_new, ws2_new);
			DBGPrintClause(SDA, firstlit);
	}
	*/
	//конфликт либо вывод второго вотча
	if (ws1_new==0 && ws2_new==0){
		if(ws2_state==NONFREE){
			//printf ("\n CONFLICT");
			return ~0;
			//return ws2_ind;
		}else{
			//return ws2_ind;
			return (ws2_var);
			//return ws2_var;
		}
	}
	//если все-таки ворочать вотчи, скорость падает на 2-5%, но уменьшается
	//количество просмотров. бред...
	if (ws2_new){
		SetBitStrided(SDA,
				ws,
				ws2_new);
		ClearBitStrided(SDA,
				ws, 
				ws2_ind);
	}
	//новый первый вотч -> возможный вывод первого вотча
	if (ws1_new){
		SetBitStrided(SDA,
				ws, 
				ws1_new);
		ClearBitStrided(SDA,
				ws, 
				firstlit);
		if(ws1_state==SOLVING){
			inspects.y+=inspects_local;
			return 0;
		}
		if(ws2_new==0 && ws2_state==NONFREE){
			//return ws1_new;
			return (ws1_var);
			//return ws1_var;
		}
	}
	return 0;
}




/*
#ifdef LITCACHE
CDF int CheckClause2(SDD,
		const litind firstlit){
	litind ws2_ind=0;
	litind ws1_new=0, ws2_new=0;
	varind ws1_var=0, ws2_var=0;
	assert(firstlit<c.lits_size);
	LIT_STATE ws1_state=NONFREE;
	LIT_STATE ws2_state=UNKNOWN;
	//uint2 current_lit=c.lits[firstlit];
	

	const uintx cachelit = c.lits[firstlit];
	litind next = cachelit.x;
	//uintx next_lit = c.lits[next];

	litind lastlit_ind = firstlit;
	int inspects_local=0;
	//prefetch через двойной буфер по вотчам и фазам только ухудшает
	//производительность
	
	bool clause_solved=false;
	varind solving_var=0;
	int solving_var_level=c.pl_size/2;
	
	
	//printf("\n %i %i %i %i", cachelit.x ,cachelit.x1,cachelit.x2,cachelit.x3);
	//int litcounter=0;
	//условия выхода из цикла: кончились литералы или BCP уже точно не будет
	//while (true){
	for(int litcounter=1; litcounter<=3; litcounter++){
		//litind current = current_lit.x;
		litind current = next;
		//uintx current_lit=next_lit;

		uint2 current_lit;

		//printf(" |> %i", current_lit.x);

		if(current==firstlit){
			break;
		}
		//next = current_lit.x;
		//next_lit = c.lits[next];


		switch(litcounter){
			case 0:{
				//current_lit.x=next_lit.x;
				//current_lit.y=next_lit.y;
				break;}
			       
			case 1:{
			       	//assert(current_lit.x==cachelit.x1);
			       	//assert(current_lit.y==cachelit.y1);
				current_lit.x=cachelit.x1;
				current_lit.y=cachelit.y1;
				break;}
			case 2:{
			       	//assert(current_lit.x==cachelit.x2);
			       	//assert(current_lit.y==cachelit.y2);
				current_lit.x=cachelit.x2;
				current_lit.y=cachelit.y2;
				break;}
			case 3:{
			       	//assert(current_lit.x==cachelit.x3);
			       	//assert(current_lit.y==cachelit.y3);
				current_lit.x=cachelit.x3;
				current_lit.y=cachelit.y3;
				break;}
		}
		next = current_lit.x;


		//assert(current<c.lits_size);
		inspects_local++;
		inspects.x++;
		//current_lit=c.lits[current];
		// TODO: оптимизировать работу со сдвигами
		uint var_ind = current_lit.y>>1;
		var_word var_container = vars[stride*bindex(var_ind*2)+shard_num];
		//var_word var_container = vars[warpIdx()][bindex(var_ind*2)];
		bool var_set= (bool)( var_container & (1 << (boffset(var_ind*2))));
		bool var_phase= (bool)(var_container & (1 << (boffset(var_ind*2+1))));
		//bool lit_phase = getLitPhase(c,current);
		bool lit_phase = !(bool)(current_lit.y&1);
		bool lit_solving = ((var_phase == lit_phase) && var_set);
		bool lit_watched = false;
		// с включением этой оптимизации производительность падает на 10% (((((
		//if(ws2_state==UNKNOWN){
		lit_watched= GetBitStrided(SDA, ws,current);
		//}

		if (lit_watched){
			if (lit_solving){
				inspects.y+=inspects_local;
				return 0;
			}
			ws2_ind=current;
			ws2_var=current_lit.y;
			if (!lit_solving && var_set){
				ws2_state = NONFREE;
			}else{
				ws2_state = FREE;
			}
			continue;
		}

		//lastlit_ind = current;

		//литерал выполняет дизъюнкт
		if (lit_solving){
			// поменять первый вотч
			ws1_new=current;
			ws1_var=current_lit.y;
			ws1_state=SOLVING;
			break;
		}
		//литерал несвободен
		if (var_set){
			continue;
		}
		//литерал свободен, второй вотч неизвестен или (известен и свободен)
		//поменять первый вотч
		if (ws1_new==0){
			ws1_new=current;
			ws1_var=current_lit.y;
			if (ws2_state!=NONFREE){
				break;
			}
			continue;
		}
		//литерал свободен, второй вотч известен и несвободен
		if (ws2_state==NONFREE){
			ws2_new=current;
			ws2_var=current_lit.y;
			break;
		}
	}
	//assert(!GetVarset(SDA, shard_num, (c.lits[ws2_ind].y)>>1));
	//assert(((c.lits[ws2_ind].y)>>1)!=((c.lits[firstlit].y)>>1));

	//конфликт либо вывод второго вотча
	if (ws1_new==0 && ws2_new==0){
		if(ws2_state==NONFREE){
			//printf ("\n CONFLICT");
			return ~0;
			//return ws2_ind;
		}else{
			//return ws2_ind;
			return (ws2_var);
			//return ws2_var;
		}
	}
	//если все-таки ворочать вотчи, скорость падает на 2-5%, но уменьшается
	//количество просмотров. бред...

	if (ws2_new){
		SetBitStrided(SDA,
				ws,
				ws2_new);
		ClearBitStrided(SDA,
				ws, 
				ws2_ind);
	}
	//новый первый вотч -> возможный вывод первого вотча
	if (ws1_new){
		SetBitStrided(SDA,
				ws, 
				ws1_new);
		ClearBitStrided(SDA,
				ws, 
				firstlit);
		if(ws1_state==SOLVING){
			inspects.y+=inspects_local;
			return 0;
		}
		if(ws2_new==0 && ws2_state==NONFREE){
			//return ws1_new;
			return (ws1_var);
			//return ws1_var;
		}
	}
	return 0;
}
#endif
*/

CDF bool checkVarConflictVars(SDD, int var)
{
	// в очереди стоят только обратные фазы присваеваемых переменых,
	// поэтому их не нужно инвертировать при проверке!
	int ind = abs(var);
	bool phase = var>0;
	
	//assert(ind!=0);
	//assert(ind<c.pl_size/2);
	if (GetVarset(SDA, ind))
	       if (GetVarphase(SDA, ind)!=phase){
		       
			if (threadWidx()==0){
			//	printf("\n1thread %d, var-varset-varphase %i-%i-%i", threadIdx.x, ind, GetVarset(SDA, shard_num, ind), GetVarphase(SDA, shard_num,ind));
			}
			return true;
		}
	return false;
	//return BCP_queue_checkfindR(SDA, shard_num, var);
}




CDF int FindNewRoot(SDD)
{
	for(int i_tp=root_trailpos; i_tp<=trail_end; i_tp++){
		const trailword current_te = trail[stride*i_tp + shard_num];
		if(isDecision(current_te) && !hasShadow(current_te)){
			return i_tp;
		}
	}
	return 0;
}


#ifdef BJ
CDF inline int getReason(trailword in){return in.reason;};

CDF void  DBGPrintTrail(SDD)
{
	printf("\n TRAIL (%i) rtp %i", trail_end, root_trailpos );
	for (int i=1; i<=trail_end; i++){
		//if(isDecision(trail[stride*i+shard_num]))
		printf(" S%i %3i:%3i-%i%i%i-r%i", shard_num, i,
				getIndex(trail[stride*i+shard_num]),
				isDecision(trail[stride*i+shard_num]),
				(int)hasShadow(trail[stride*i+shard_num]),
				trail[stride*i+shard_num].var&1,
				getReason(trail[stride*i+shard_num]));
	}
	printf("\n");
}

CDF int FillReasonBitmapFromLit(SDD, var_word* reason_bitmap, 
		const litind startlit, const varind var=0)
{
	litind current_lit=startlit;
	//if (shard_num==583) DBGPrintClause(SDA, startlit);
	//printf("\n startlit %i", startlit);
	//if (shard_num==583)printf("\n start fill reason bitmap for var %i", c.lits[current_lit].y>>1);
	int count=0;
	do{
	//printf("\n currentlit %i", current_lit);
		if(c.lits[current_lit].y>>1 != var){
			SetBitSimple(SDA, reason_bitmap, c.lits[current_lit].y>>1);
			count++;
			//if (shard_num==583)printf("\n fill reason bitmap  %i", c.lits[current_lit].y>>1);
		}
		current_lit=c.lits[current_lit].x;
	}while(current_lit!=startlit);
	return count;
}

CDF void RestoreReasonFromShadow(SDD, var_word* reason_bitmap,
		const int reason_list_start)
{
	for(int i=reason_list_start; i<shadow_reasons_end; i++){
		assert(reason_list_start<=shadow_reasons_end);
		assert(reason_list_start!=0);
		assert(reason_list_start>0);
		assert(shadow_reasons_end>0);
		assert((shard_num*HIDDEN_REASONS_SIZE+i)>=0);
		const shadowword current = shadow_reasons[shard_num*HIDDEN_REASONS_SIZE+i];
		assert (current<(c.pl_size/2));
		assert (current>0);
		SetBitSimple(SDA, reason_bitmap, current);
	}
}

CDF int StoreReasonToShadow(SDD, var_word* reason_bitmap)
{
	const int reason_list_start = shadow_reasons_end;
	for(int i=0; i<c.pl_size/2; i++){
		if (GetBitSimple(SDA, reason_bitmap, i)){ 
			shadow_reasons[shard_num*HIDDEN_REASONS_SIZE+shadow_reasons_end]=i;
			assert(shadow_reasons_end<HIDDEN_REASONS_SIZE);
			shadow_reasons_end++;
		}
	}
	return reason_list_start;
}

CDF int StoreReasonToShadowFast(SDD, var_word* reason_bitmap,
		const int lits_count)
{
	const int reason_list_start = shadow_reasons_end;
	int lits_found=0;
	for(int i=trail_end; i>0; i--){
		const int current_ind = getIndex(trail[stride*i + shard_num]);
		if (GetBitSimple(SDA, reason_bitmap, current_ind)){ 
			assert(shadow_reasons_end<HIDDEN_REASONS_SIZE);
			if (shadow_reasons_end>=HIDDEN_REASONS_SIZE){
				printf("\n REASONS OVERFLOW shard %i",
						shard_num);
				asm("trap;");
			}
			shadow_reasons[shard_num*HIDDEN_REASONS_SIZE+shadow_reasons_end]=current_ind;
			shadow_reasons_end++;
			lits_found++;
			if(lits_found==lits_count){
				//FIXME неправильно считаем, нужно
				//учитывать и теневые. так что пока
				//считаем все
				//break;
			}
		}
	}
	return reason_list_start;
}

CDF inline int BackjumpFast(SDD, const litind conflict_lit)
{
	var_word reason_bitmap[(VARS_SIZE)/(sizeof(var_word)*8)];
	for (int i=0; i<(VARS_SIZE)/(sizeof(var_word)*8); i++){
		reason_bitmap[i]=0;
		//assert(reason_bitmap[i]==0);
	}
	bool stop_backtrack = false; 
	int lits_count=0;
	lits_count += FillReasonBitmapFromLit(SDA, reason_bitmap, conflict_lit);

	//if (shard_num==583) printf("\n Start BJ");
	// откатываться можно не дальше 1-го родителя !!!!
	int shadow_handle = ~0;
	do{
		trailword current = trail[stride*trail_end+shard_num];
		while (!isDecision(current)){
		// очищаем выведенные переменные
			if (!stop_backtrack){
				if (GetBitSimple(SDA, reason_bitmap, getIndex(current))){
				//if (false){ //BACKTRACKING
					//printf("\n trail end %i", trail_end);
					assert(trail_end>root_trailpos);
					assert(getIndex(current)<c.pl_size/2);
					assert(getReason(current)!=0);
					assert(getReason(current)>0);
					assert(getReason(current)<c.lits_size);
					assert(!isDecision(current));
					assert(getIndex(current)>getIndex(trail[stride*root_trailpos+shard_num]));
					assert(current.reason!=0);
					assert(current.reason!=~0);
					assert(!hasShadow(current));
					
					//if (shard_num==583)printf("\n start fill reason bitmap for var %i", getIndex(current));
					lits_count+= FillReasonBitmapFromLit(SDA,
							reason_bitmap,
							getReason(current),
							getIndex(current));
					lits_count--;
					ClearBitSimple(SDA,
							reason_bitmap,
							getIndex(current));
				}
			}
			//ClearVar(SDA, getIndex(current));
			ClearVarset(SDA, getIndex(current));
			trail_end--;
			current = trail[stride*trail_end+shard_num];
		}
		
		// переключаем переменную уровня решения
		assert(isDecision(current));

		assert(getReason(current)<=HIDDEN_REASONS_SIZE);
		assert(getReason(current)>=0);
		decision_var_index=getIndex(current);
		//if(shard_num==10){ printf("%u ", decision_var_index); }

		
		/*
		if (!hasShadow(current)){
				stop_backtrack=true;
				shadow_handle=1;
		}
		*/
		
		//if (shard_num==583) printf("  %i-%i-%i", decision_var_index, GetBitSimple(SDA, reason_bitmap, decision_var_index), hasShadow(current));
		//if (!GetBitSimple(SDA, reason_bitmap, decision_var_index ) && ((current.var&1)==1)){ printf("\n BACKJUMP");}
		ClearVarset(SDA, decision_var_index);

		trail_end--;
		//дошли до корневой переменной 
		//if (decision_var_index==getIndex(trail[stride*root_trailpos+shard_num])){
		if ((trail_end+1)==root_trailpos){
			stop_backtrack=true;
			 // пытаемся залочить корень 
			 if((atomicCAS((uint*)&thread_state[shard_num],
						 ROOT_READY,
						 ROOT_NOT_READY) ==
					 ROOT_READY)){
				// удалось залочить. Проверяем,
				// требовалось ли откатываться выше
			 	if(hasShadow(current)){
					// требовалось откатиться выше,
					// но уже дошли до корня.
					// Значит, откат невозможен
					shadow_handle=~0;
				}
			 	if(!GetBitSimple(SDA, reason_bitmap, decision_var_index)){
					// переменная не является
					// причиной конфликта и должна
					// быть пропущена.
					// Откат невозможен.
					shadow_handle=~0;
				}
				shadow_handle=0;
			 }else{
				// залочить не удалось. значит откат
				// невозможен
				shadow_handle=~0;
			 }

		}else{
			// если текущая переменная уровня решения является одной
			// из причин конфликта:
			if (GetBitSimple(SDA, reason_bitmap, decision_var_index)){
			//if (true){ // BACKTRACKING
				if(hasShadow(current)){
					// есть тень, то есть уже перещелкивали - восстанавливаем теневые причины
					RestoreReasonFromShadow(SDA, reason_bitmap, getReason(current));
				}else{
					// нет тени, значит нашли самую нижнюю
					// еще не перещелкивавашуюся
					// переменную уровня решения из
					// множества переменных,
					// ответственных за конфликт. Сохраняем
					// текущие теневые причины и заканчиваем
					// откат.
					stop_backtrack=true;
					shadow_handle = StoreReasonToShadowFast(SDA, reason_bitmap, lits_count);
					//shadow_handle = StoreReasonToShadow(SDA, reason_bitmap);
					//shadow_handle = 1; //BACKTRACKING
				}
			}
			// спиливаем уровень в массиве теневых причин так, чтобы
			// он соответствовал текущей переменной уровня решения
			if(hasShadow(current) && !stop_backtrack){
				shadow_reasons_end=getReason(current);
			}
		}
		//повторяем процесс 
	} while (!stop_backtrack);
	BCPQueueClear(SDA);
	return shadow_handle;
}
#endif
/*
CDF inline uint backtracking(SDD, const litind conflict_lit){
	bool backtrack_impossible=false;
	bool stop_backtrack = (GetVarphase(SDA, decision_var_index)==DEFAULT_PHASE);
	do{
		trailword current = trail[stride*trail_end+shard_num];
		// меткой переменной уровня решения является её знак
		while (!isDecision(current)){
			// очищаем выведенные переменные
			ClearVar(SDA, getIndex(current));
			--trail_end;
			current = trail[stride*trail_end+shard_num];
		}
		
		// переключаем переменную уровня решения
		assert(isDecision(current));
		decision_var_index=isDecision(current);

		//перещелкиваем причину
		if (GetVarphase(SDA, decision_var_index)==DEFAULT_PHASE){
			stop_backtrack=true;
		}
		ClearVar(SDA, decision_var_index);

		trail_end--;
		// уже дошли до переменной верхнего уровня, но даже и не собираемся останавливаться
		if ((decision_var_index==top_decision_var_index) && !stop_backtrack){
			stop_backtrack=true;
			backtrack_impossible=true;
		}
		//повторяем процесс 
	} while (!stop_backtrack);
	BCPQueueClear(SDA);
	return (uint)backtrack_impossible;
}
*/

CDF inline int findNextFreeVar(SDD, int var)
{
	while (GetVarset(SDA, abs(var))){
		++var;
		}
	//assert(var<c.pl_size/2);
	return var;
}

CDF void DefineDecisionVar(SDD)
{
	//определение номера новой переменной решения
	//assert(decision_var_index<c.pl_size/2);
	//while(++decision_var_index < c.pl_size/2){
	//	if(!GetVarset(SDA, decision_var_index)){
	for (int i=1; i< c.pl_size/2; i++){
		if(!GetVarset(SDA, vars_heap[stride*i+shard_num])){
			decision_var_index=vars_heap[stride*i+shard_num];
			//__threadfence();
			break;
		}
	}
	//assert(!searchTrail(SDA, shard_num, decision_var_index));
	/*
	if ((int)decision_var_index>= (int)c.pl_size/2){
		printf("\n var_index / pl_size/2 :   %i / %i",decision_var_index , c.pl_size/2);
	}
	assert(decision_var_index<c.pl_size/2);
	*/
}
