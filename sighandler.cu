#include <stdio.h>
#include <stdlib.h>
#include <signal.h>
// State variables for sigint_handler                                                                                                                                                                                                        
extern int no_sigint;
int no_sigint = 1;
extern int interrupts;
int interrupts = 0;
/* Catches signal interrupts from Ctrl+c.
If 1 signal is detected the simulation finishes the current frame and exits in a clean state. If Ctrl+c is pressed again it terminates the application without completing writes to files or calc
ulations but deallocates all memory anyway.
*/
void sigint_handler (int sig)
{
  if (sig == SIGINT)
    {
      interrupts += 1;
      std::cout << std::endl << "Aborting loop.. finishing frame." << std::endl;
      no_sigint = 0;

      if (interrupts>=2)
      {
        std::cerr << std::endl << "Multiple Interrupts issued: Clearing memory and Forcing immediate shutdown!" << std::endl;
        free_mem(); // write a function to free dynamycally allocated memory
        int devCount;
        hipGetDeviceCount(&devCount);
        for (int i=0;i<devCount;++i) {hipSetDevice(i); cudaResetDevice();}
        exit(9);
      }
    }
}
