#include "hip/hip_runtime.h"
#include "solver.h"
#include <signal.h>
#include <unistd.h>
#include "solver_cuda.h"
#include <stdio.h>
#include "cuda_error_check.cu"

// вставляем флаг CUDA перед определениями общих функций
#define CDF __device__
#include "solver_common.cu"

#define lane threadWidx()
#define SPIN_LIMIT (1<<25)
#define RNG_CYCLES 200


volatile sig_atomic_t e_flag = 0;

void SleepSomeTime(unsigned int ns)
{
	struct timespec t;
	t.tv_sec = 0;
	t.tv_nsec = (long)ns;
	nanosleep(&t, NULL);
}

__device__ uint GetRandomNumber(uint seed)
{
	for (int i=0; i<RNG_CYCLES; i++){
		//seed=(seed<<3)^(seed>>1);
		seed=(seed<<1)^((seed>>31)&1)^((seed>>29)&1)^((seed>>25)&1)^((seed>>24)&1);
	}
	return seed;
}
void INTHandler(int sig)
{
	e_flag = 1;
	/*
	signal(sig, SIG_IGN);
	const int stop_mark=1;
	hipMemcpyToSymbolAsync(HIP_SYMBOL(kernel_stop_signal), (void*)
				&stop_mark,
				sizeof(int), 0,
				hipMemcpyHostToDevice, signal_stream);
	signal(SIGUSR1, INTHandler);
	*/
}

texture<int, hipTextureType1D, hipReadModeElementType> texPl;

hipStream_t solver_kernel_stream;
hipStream_t signal_stream;





__device__ int shnums_tmp[1<<14];

__device__ int complete_blocks=0;
__device__ int task_complete=0;
__device__ int4 shards_complete={0,0,0,0};
__device__ int4 warps_complete={0,0,0,0};
__device__ volatile int kernel_stop_signal=0; // флаг того, что нашлось решение.
//int4* warps_complete=NULL;


__device__ litind propagate_thread(SDD, 
		const int stall = DEFAULT_VOTE_THRESHOLD,
		const bool RemoveWatches = false)
{
	int i=0;
	varind var_index = 0 ;
	int pl_index = 0;
	int start = 0;
	int end = 0;
	litind conflict = 0;
	bool var_done=true;
	//while(!isBCPQueueEmpty(SDA, shard_num)){
	//while(__any(!isBCPQueueEmpty(SDA))){
	//while(__all(!isBCPQueueEmpty(SDA))){
	while(!isBCPQueueEmpty(SDA)){
		int nonempty_bcp_count=__popc(__ballot(!isBCPQueueEmpty(SDA)));
		// Голосование!
		if(!((nonempty_bcp_count > stall) || __any(stall_count!=0))){
			stall_count=0;
			if(!isBCPQueueEmpty(SDA) && !conflict && i!=end){
				stall_count=i;
				stalled_var_index=var_index;
				//pl_index = stalled_pl_index;
				stalled_start = start;
				stalled_end = end;
			}
			if(!isBCPQueueEmpty(SDA) && !conflict && i==end){
				stall_count=(-1);
			}
			break;
		}
		if(isBCPQueueEmpty(SDA)){
			continue;
			//break;
		}
		//берем очередную переменную из очереди (переменная уже означена)
		if(var_done){
			if(stall_count>0){
				var_index = stalled_var_index;
				start = stalled_start;
				end = stalled_end;
				i = stall_count;
				assert (stall_count>=start);
				assert (stall_count<end);
				stall_count=0;
			}else{
				var_index = getIndex(trail[stride*BCP_queue_front+shard_num]);
				assert(var_index<c.pl_size/2);
				pl_index = HVarToVar((1-2*GetVarphase(SDA, var_index))*var_index);
				start = c.pl[pl_index-1];
				end = c.pl[pl_index];
				i = start;
				wsword = ws[stride*bindex(i)+shard_num];
			}
			var_done=false;
		}
		while (!(wsword & (1 << (boffset(i))))&& (i<end)){
			i++;
			if(boffset(i)==0 && (i<end)){
				wsword = ws[stride*bindex(i)+shard_num];
			}
		}
		

		
		if (i==end){
			BCPQueuePopFront(SDA);//удаляем переменную из очереди
			//printf("\n STOP %i", var_index  );
			var_done=true;
			continue;
		}
#ifdef LITCACHE
		int bcpvar = CheckClause2(SDA, i);
#else
		int bcpvar = CheckClause(SDA, i);
#endif
		if (bcpvar==~0){
			conflict = (litind) i;
			BCPQueueClear(SDA);
			continue;
		}
		if (bcpvar!= 0){
			//assert(clauseUnit(SDA, i));
			assert((bcpvar>>1)>getIndex(trail[stride*root_trailpos+shard_num]));
			SetVar(SDA, bcpvar, (litind)i );

		}else{
			//assert(!clauseUnit(SDA, i));
		}
			//assert(!clauseEmpty(SDA, i));
		i++;
		if(boffset(i)==0 && (i<end)){
			wsword = ws[stride*bindex(i)+shard_num];
		}
	}
	if(isBCPQueueEmpty(SDA)){
		stall_count=0;
	}
	return conflict;
}


__device__ void CopyRootStateFromThread(SDD, const int src_shard_num)
{
	shadow_reasons_end=1;
	// очищаем текущее состояние
	while(trail_end>=1){
		trailword current = trail[stride*trail_end+shard_num];
		ClearVarset(SDA, getIndex(current));
		--trail_end;
	}
	BCPQueueClear(SDA);

	//копируем данные с донора
	root_trailpos=employer_transfer_vars[src_shard_num].root_trailpos;
	assert((getIndex(trail[stride*root_trailpos+src_shard_num]))<c.pl_size/2);
	for (int i=1; i<root_trailpos;i++){
		assert(((trail[stride*i+src_shard_num].var&VARINDEX_MASK)>>1)<c.pl_size/2);
		SetVar(SDA, trail[stride*i+src_shard_num].var&VARINDEX_MASK, 0,1);
	}
	//устанавливаем верхнюю корневую переменную
	SetVar(SDA, ((trail[stride*root_trailpos+src_shard_num].var)&VARINDEX_MASK)^1, 0,1);
	trail[stride*BCP_queue_front+shard_num].var|=SEEN_FLAG;
	decision_var_index = getIndex(trail[stride*root_trailpos+shard_num]); 
	assert(decision_var_index<c.pl_size/2);
	//if (shard_num==1) printf("\n DVI %i prev trail %i",decision_var_index, getIndex(trail[stride*(trail_end)+shard_num]));

	state=FORCED_DECISION;
}

__device__ void ShareWorkInMyWarp(SDD)
{
	volatile threadstate* tgt_lockp = &thread_state[shard_num];
	if (state==UNSAT) assert(thread_state[shard_num]!=ROOT_READY);
	const threadstate old_lockdata = (state==UNSAT) ? ROOT_NOT_READY:ROOT_READY;
	threadstate new_lockdata = old_lockdata;
	const threadstate thread_lock_signature= ROOT_NOT_READY; 
	__threadfence();
	const bool thread_locked=(atomicCAS((uint*)tgt_lockp, old_lockdata, thread_lock_signature) == old_lockdata);
	const bool iam_employer = ((state==BCP_OK ||
			state==FORCED_DECISION ||
			state==DEFAULT_DECISION ||
			state==BCP_STALL) &&
			!hasShadow(trail[stride*root_trailpos+shard_num]) && 
			thread_locked);
	const bool iam_jobless = ((state==UNSAT)&&thread_locked);
	const uint jobless_map = __ballot(iam_jobless);
	const uint employer_map= __ballot(iam_employer);
	const int jobless_count= __popc(jobless_map);
	const int employer_count= __popc(employer_map);
	if (jobless_count>0 && employer_count>0){
		//const int employer_count= WARP_SIZE - jobless_count;
		const int jobless_id = __popc(jobless_map&lanemask_lt());
		assert (jobless_id<WARP_SIZE);
		//const int employer_id = lane - jobless_id;
		int employer_id = __popc(employer_map&lanemask_lt());
		__shared__ int employer_id_array[DEFAULT_BLOCKSIZE];
		__shared__ int employer_root_trailpos_array[DEFAULT_BLOCKSIZE];
		__shared__ int employer_trailend_array[DEFAULT_BLOCKSIZE];
		
		// Перемешиваем номера, чтобы более-менее равномерно
		// работа внутри варпа распределялась
		if (iam_employer && employer_count>1){
			uint rng_state=GetRandomNumber(clock());
			int shift=abs(((int)rng_state)&31);
			employer_id=(employer_id+shift)%employer_count;
		}
		if(iam_employer && employer_id<jobless_count){

			// мне помогут
			assert(!iam_jobless);
			employer_id_array[employer_id+warpIdx()*WARP_SIZE] = lane;
			employer_root_trailpos_array[employer_id+warpIdx()*WARP_SIZE] = root_trailpos;
			employer_trailend_array[employer_id+warpIdx()*WARP_SIZE] = trail_end;
			//перемещаем корень
			trail[stride*root_trailpos+shard_num].var=trail[stride*root_trailpos+shard_num].var | SEEN_FLAG;
			new_lockdata=ROOT_NOT_READY;
			++inspects.z;
		}
		const int my_employer = employer_id_array[jobless_id+warpIdx()*WARP_SIZE];
		assert (my_employer<WARP_SIZE);
		const int employer_shard_num =
			my_employer + 
			warpIdx()*WARP_SIZE +
			blockIdx.x*blockDim.x;
		
		if(iam_jobless && jobless_id<employer_count){
			assert(!iam_employer);
			//я помогу
			//копируем данные с хозяина
			root_trailpos = employer_root_trailpos_array[jobless_id+warpIdx()*WARP_SIZE];
			trail_end = employer_trailend_array[jobless_id+warpIdx()*WARP_SIZE];
			BCPQueueClear(SDA);


			// Сбрасываем массив теневых причин
			shadow_reasons_end=1;
			//TODO: убрать лишнее копирование следа
			for (int i=0; i<c.pl_size/2;i++){
				trail[stride*i+shard_num]=trail[stride*i+employer_shard_num];
			}
			/*
			for (int i=0; i<c.pl_size/WORD_SIZE+1;i++){
				vars[stride*i+shard_num]=vars[stride*i+employer_shard_num];
			}
			*/
		}
		// кооперативно копируем с нанимателей на
		// рабочих данные через промежуточные переменные
		// в общей памяти
		//TODO: вынести в отдельную функцию копирования
		for (int i=0; i<c.lits_size/WORD_SIZE+1;i++){
			__shared__ var_word tmp[DEFAULT_BLOCKSIZE];
			if((employer_id<jobless_count) && iam_employer){
				tmp[employer_id+warpIdx()*WARP_SIZE]=ws[stride*i+shard_num];
			}
			if((jobless_id<employer_count) && iam_jobless){
				ws[stride*i+shard_num]=tmp[jobless_id+warpIdx()*WARP_SIZE];
			}
		}
		for (int i=0; i<c.pl_size/WORD_SIZE+1;i++){
			__shared__ var_word tmp[DEFAULT_BLOCKSIZE];
			if((employer_id<jobless_count) && iam_employer){
				tmp[employer_id+warpIdx()*WARP_SIZE]=vars[stride*i+shard_num];
			}
			if((jobless_id<employer_count) && iam_jobless){
				vars[stride*i+shard_num]=tmp[jobless_id+warpIdx()*WARP_SIZE];
			}
		}
		if(iam_jobless && jobless_id<employer_count){
			// очищаем лишние переменные, скопированные с хозяина
			while(trail_end>=root_trailpos){
				trailword current = trail[stride*trail_end+shard_num];
				ClearVarset(SDA, getIndex(current));
				--trail_end;
			}
			//top_decision_var_index=decision_var_index;

			//if (shard_num==1) printf("\n DVI %i prev trail %i",decision_var_index, getIndex(trail[stride*(trail_end)+shard_num]));
			assert(decision_var_index<c.pl_size/2);
			BCPQueueClear(SDA);
			// устанавливаем переменную верхнего уровня решения в фазу, обратную той, что была у хозяина
			SetVar(SDA, ((trail[stride*root_trailpos+employer_shard_num].var)&VARINDEX_MASK)^1, 0,1);
			trail[stride*BCP_queue_front+shard_num].var|=SEEN_FLAG;
			decision_var_index = getIndex(trail[stride*root_trailpos+shard_num]); 
			state=FORCED_DECISION;
			
			/*
			if (my_employer==3 || my_employer==5 || shard_num==3 || shard_num == 5)
					printf("\n block %i lane %i employs lane %i : erv %i wdvi %i ",
					blockIdx.x, my_employer,
					lane,
					getIndex(sd.trail[sd.stride*root_trailpos+employer_shard_num]),
					decision_var_index) ;
					*/
					
			atomicAdd(&shards_complete.x,1);
			new_lockdata=ROOT_NOT_READY;
		}
	}
	__threadfence();
	if (thread_locked){
		thread_state[shard_num]=new_lockdata;
	}
}



__device__ bool TryExchangeWithThread(SDD, const int tgt_thread)
{
	volatile threadstate* tgt_lockp=  &thread_state[tgt_thread];
	const threadstate old_lockdata = thread_state[tgt_thread];
	const threadstate my_steal_signature= (shard_num | ROOT_STOLEN); 

	// проверим готовность партнера
	if((old_lockdata&THREADSTATE_MASK)!=ROOT_READY){
		//printf("\n thread %i old lockdata %x", shard_num, old_lockdata);
		return false;
	}
	// предложим обмен
	if(!atomicCAS((uint*)tgt_lockp, old_lockdata, my_steal_signature) == old_lockdata){
		// кто-то перехватил предложение
		assert(false);
		return false;
	}
	__threadfence();
	return true;
}

__device__  void PermutateNums(int multi, const float num_permutations = 2){
	//return;
	//uint rng_state=GetRandomNumber(clock()^shard_num);
	//uint rng_state=GetRandomNumber(123456789^(shard_num>>5));
	uint rng_state=GetRandomNumber(123456789);
	//uint rng_state=GetRandomNumber(123456789);
	for(int i=0; i<multi; i++){
		shnums_tmp[i]=i;
	}
	for(int i=0; i<multi; i++){

		rng_state=GetRandomNumber(rng_state);
		int x_address=  ((float)(rng_state&0x0fffffff) /
				(float)0x0fffffff) * (multi-1);
		assert(x_address > 0);
		assert(x_address < multi);
		rng_state=GetRandomNumber(rng_state);
		int y_address=  ((float)(rng_state&0x0fffffff) /
				(float)0x0fffffff) * (multi-1);
		assert(y_address > 0);
		assert(y_address < multi);
		varind x = shnums_tmp[x_address];
		varind y = shnums_tmp[y_address];
		shnums_tmp[x_address] = y;
		shnums_tmp[y_address] = x;
	}
	//for(int i=1; i<multi; i++){ printf("\n SHNUM %u",shnums_tmp[i]); }
}

__device__  void PermutateVarsShard(SDD, const float num_permutations = 2){
	//return;
#ifndef EQUAL_THREADS_IN_WARP
	uint rng_state=GetRandomNumber(123456789^shard_num);
#else
	uint rng_state=GetRandomNumber(123456789^(shard_num>>5));
#endif
	//uint rng_state=GetRandomNumber(123456789^shard_num);
	//uint rng_state=GetRandomNumber(123456789);
	for(int i=1; i<c.pl_size/2*num_permutations; i++){
		rng_state=GetRandomNumber(rng_state);
		int x_address= 1+ ((float)(rng_state&0x0fffffff) /
				(float)0x0fffffff) * (c.pl_size/2-1);
		assert(x_address > 0);
		assert(x_address < c.pl_size/2);
		rng_state=GetRandomNumber(rng_state);
		int y_address= 1+ ((float)(rng_state&0x0fffffff) /
				(float)0x0fffffff) * (c.pl_size/2-1);
		assert(y_address > 0);
		assert(y_address < c.pl_size/2);
		varind x = vars_heap[stride*x_address+shard_num];
		varind y = vars_heap[stride*y_address+shard_num];
		vars_heap[stride*x_address+shard_num] = y;
		vars_heap[stride*y_address+shard_num] = x;
	}
	/*
	printf("\n RNG %u",rng_state);
	for(int i=0; i<100; i++){
	if (shard_num==0)
		printf("\n var %i  %i", i, vars_heap[stride*i+shard_num]);
	if (shard_num==1)
		printf("\n var %i  %i", i, vars_heap[stride*i+shard_num]);
	}
	*/
	
#ifndef NDEBUG
	for(int i=0; i<c.pl_size/2; i++){
		bool varfound=false;
		for(int m=0; m<c.pl_size/2; m++){
			if(vars_heap[stride*m+shard_num]==i){
				varfound=true;
				break;
			}
		}
		assert(varfound);
	}
#endif
}
__global__ void __launch_bounds__(DEFAULT_BLOCKSIZE,8) PreprocessR(
		gpusolverdata sd,
		shadowword* shadow_reasons,
		volatile threadstate* thread_state,
	       	gpusolverdata sd_temp,
		int first_trail_end,
		int first_BCP_queue_front,
		int* partial_warps,
		volatile solver_internal_vars* employer_transfer_vars,
		const int subproblem=0)
{
	const int inspects_limit=1<<30;

	int shard_num = threadIdx.x+blockIdx.x*blockDim.x;
	//printf("\n Preprocess START  ");

	//uint trail_end = first_trail_end;
	
	//if(threadIdx.x==0){
	/*
	for(int i=0; i<WARP_SIZE;i++){
		partial_warps[i]=0;
	}
	*/
	//sd.conflict_count_old[shard_num]=555555;
	//sd.stall_count[shard_num]=0;
	sd.out[shard_num]=BCP_OK;
	sd.inspects[shard_num].x=0;
	sd.inspects[shard_num].y=0;
	sd.inspects[shard_num].z=0;
	sd.inspects[shard_num].w=0;
	sd.decision_var_index[shard_num]=0;
	sd.trail_end[shard_num]=first_trail_end;
	/*
	for(int i=0;i<BCP_QUEUE_SIZE;i++){
		sd.BCP_queue[shard_num][i]=444444;
	}
	*/
	//__threadfence();

	varind* vars_heap=sd.vars_heap;

	shards_complete.x=0;
	shards_complete.y=0;
	shards_complete.z=0;
	shards_complete.w=0;
	warps_complete.x=0;
	warps_complete.y=0;
	warps_complete.z=0;
	warps_complete.w=0;
	for (int i=0; i<sd.c.pl_size/2;i++){
		//assert(abs((int)(sd_temp.trail[i]))<sd.c.pl_size/2);
		sd.trail[sd.stride*i+shard_num]=sd_temp.trail[i];
		//if(shard_num==26){printf("\n tv %i %i", i, getIndex(sd_temp.trail[i]));} }
		sd.vars_heap[sd.stride*i+shard_num]=i;
	}
	
	for (int i=0; i<sd.c.pl_size/WORD_SIZE+1;i++){
		sd.vars[sd.stride*i+shard_num]=sd_temp.vars[i];
	}

	for (int i=0; i<sd.c.lits_size/WORD_SIZE+1;i++){
		//sd.ws[(sd.c.lits_size/WORD_SIZE+1)*shard_num+i]=sd_temp.ws[i];
		sd.ws[sd.stride*i+shard_num]=sd_temp.ws[i];
	}

	int4 inspects=sd.inspects[shard_num];
	RESULT state=sd.out[shard_num];
	int trail_end=sd.trail_end[shard_num];
	int decision_var_index=sd.decision_var_index[shard_num];
	//int BCP_queue_front=sd.BCP_queue_front[shard_num];
	int BCP_queue_front=first_BCP_queue_front;
	int stall_count=0;
	int stalled_start, stalled_end;
	varind stalled_var_index;
	var_word wsword;

	int shadow_reasons_end = 1;

	//bool move_root=false;
	//int root_trailpos=trail_end;
	int root_trailpos=0;
	sd.trail[sd.stride*root_trailpos+shard_num].var=0;
	const int stride=sd.stride;
	Rcnf& c = sd.c;
	var_word* &ws=sd.ws;
	var_word* &vars=sd.vars;
	trailword* &trail=sd.trail;


	//assert(!verifytrail(sd, shard_num));
	//BCPQueueClear(SDA);
	
	//printf("\n TRAILEND_GPU %i",trail_end);
	// Дефолтная фаза
	for (int i=0; i<sd.c.pl_size/2;i++){
		if(!GetVarset(SDA, i)){
	 		SetVarphase(SDA, i);
	 		//ClearVarphase(SDA, i);
			}
	}
/*	
	for(int j=0;j<13;j++){
		varind nfv = findNextFreeVar(SDA, j+1);
		if ((subproblem>>j)&1){
			nfv = findNextFreeVar(SDA, nfv+1);
		}
		SetVar(SDA, HVarToVar(nfv));
		bool conflict=propagate_thread(SDA, 0);
		BCPQueueClear(SDA);
		if (conflict){
			state = UNSAT;
			//printf("\n Prop Stop");
			atomicAdd(&shards_complete.y,1);
			break;
		} else{
			if (trail_end==(c.pl_size/2-1)){
				state = SAT;
				atomicAdd(&shards_complete.x,1);
				//printf("\n %i Trail end %i", shard_num, trail_end);
				break;
			}
		}
	}
*/	
	
#ifdef PERMUTATE_VARS_ORDER
	PermutateVarsShard(SDA);
#endif
	
	for(uint j=0;j<sd.multi;j++){
	//for(uint j=0;j<0;j++){
	//for(uint j=0;j<5;j++){
#ifndef EQUAL_THREADS_IN_WARP
		int nfv = findNextFreeVar(SDA, 1)*(1-2*((shard_num>>j)&1));
#else
		int nfv = findNextFreeVar(SDA, 1)*(1-2*((shard_num>>(sd.multi-j+4))&1));
#endif 
		//int nfv = findNextFreeVar(SDA, 1)*(1-2*((shard_num>>(sd.multi-j-1))&1));
		
		/*	
		varind nfv = findNextFreeVar(SDA, j+1);
		if ((shard_num>>j)&1){
			nfv = findNextFreeVar(SDA, nfv+1);
		}
		*/
		
		
		//assert(!searchTrail(sd, shard_num, nfv ));
		//nfv = 32;
		//if(threadWidx()==0) printf("\n Thread %i nfv %i", threadIdx.x, nfv);
		//BCP_queue_pushbackR(sd, shard_num, nfv);
		//bool RemoveWatches=true;
		bool RemoveWatches=false;
		SetVar(SDA, HVarToVar(nfv),0,1);
		//__syncthreads();
		//BCP_queuePrint(sd, shard_num);
		
		litind conflict = propagate_thread(SDA, 0, RemoveWatches);
		BCPQueueClear(SDA);
		if (conflict!=0){
			state = UNSAT;
			//printf("\n Prop Stop %i", threadIdx.x);
			atomicAdd(&shards_complete.y,1);
			break;
		} else{
			for(int i=2; i<c.lits_size; i++){
				if(c.lits[i].x!=NO_LITIND){
					if(clauseUnit(SDA, i)){
						if(shard_num==3){
							DBGPrintClause(SDA, i);
						}
						assert(clauseEmpty(SDA, i));
						assert(false);
					}
				}
			}
			//printf("\n %i Trail end %i", shard_num, trail_end);
			if (trail_end==(c.pl_size/2-1)){
				state = SAT;
				atomicAdd(&shards_complete.x,1);
				//printf("\n %i Trail end %i", shard_num, trail_end);
				break;
			}
		}
	}
/*
	sd.inspects[shard_num].x=0;
	sd.inspects[shard_num].y=0;
	sd.inspects[shard_num].z=0;
	sd.inspects[shard_num].w=0;
	*/
	sd.inspects[shard_num].x+=inspects.x;
	sd.inspects[shard_num].y+=inspects.y;
	sd.inspects[shard_num].z+=inspects.z;
	sd.out[shard_num]=state;
	sd.trail_end[shard_num]=trail_end;

	sd.decision_var_index[shard_num]=decision_var_index;
	sd.BCP_queue_front[shard_num]=BCP_queue_front;
	
	thread_state[shard_num]=ROOT_NOT_READY;
	//printf("\n Thread %i PP result %i , inspects %i",threadIdx.x, result, sd.inspects[shard_num]);
	//printf("\n %i Trail end %i", shard_num, trail_end);
	//PermutateVarsShard(SDA);
	//shnums_tmp[shard_num]=shard_num; 
	if (shard_num==0) PermutateNums(1<<sd.multi); 

}

__device__ void StoreLocalVars(SDD)
{
	employer_transfer_vars[shard_num].state=state;
	employer_transfer_vars[shard_num].trail_end=trail_end;
	employer_transfer_vars[shard_num].root_trailpos=root_trailpos;
	employer_transfer_vars[shard_num].BCP_queue_front=BCP_queue_front;
	employer_transfer_vars[shard_num].decision_var_index=decision_var_index;
}
__device__ void LoadLocalVars(SDD)
{
	state=employer_transfer_vars[shard_num].state;
	trail_end=employer_transfer_vars[shard_num].trail_end;
	root_trailpos=employer_transfer_vars[shard_num].root_trailpos;
	BCP_queue_front=employer_transfer_vars[shard_num].BCP_queue_front;
	decision_var_index=employer_transfer_vars[shard_num].decision_var_index;
}


__device__ bool CheckSAT(SDD)
{
	assert(isBCPQueueEmpty(SDA));
	if (trail_end==(c.pl_size/2-1)){
		assert(!verifytrail(SDA));
		assert(!DBGVerifyVars(SDA));
		for(int i=2; i<c.lits_size; i++){
			if(c.lits[i].x!=NO_LITIND){
				if(!clauseSolved(SDA, i)){
					assert(clauseEmpty(SDA, i));
					if(shard_num==1){
						DBGPrintClause(SDA, i);
					}
					assert(false);

				}
			}
		}
			
		//atomicAdd(&shards_complete.x,1);
		return true;
	}
	return false;
}
__device__ void MakeNewDecision(SDD)
{
	assert(isBCPQueueEmpty(SDA));
	DefineDecisionVar(SDA);
	//ставим в очередь BCP переменную уровня решения
	assert(decision_var_index<c.pl_size/2);
	SetVar(SDA, HVarToVar((int)decision_var_index* (-1+2*(GetVarphase(SDA, decision_var_index)))), 0, 1);
	
	//assert(!searchTrail(SDA, shard_num, decision_var_index[shard_num]));
	if(thread_state[shard_num]==ROOT_NOT_READY){
		trail[stride*root_trailpos+shard_num].var=trail[stride*root_trailpos+shard_num].var | SEEN_FLAG;
		//переставляем корень
		int new_root_trailpos = FindNewRoot(SDA);
		assert(new_root_trailpos!=0);
		root_trailpos = new_root_trailpos;
		StoreLocalVars(SDA);
		thread_state[shard_num]=ROOT_READY;
		++inspects.w;
	}
	assert(!hasShadow(trail[stride*root_trailpos+shard_num]));
	assert( isDecision(trail[stride*root_trailpos+shard_num]));
}
__device__ bool StealWorkFromOtherWarps(SDD)
{
#ifdef WORK_SHARE
	uint rng_state=GetRandomNumber(clock());
	//uint rng_state=GetRandomNumber(my_warp_num());
	//int donor_warp=(stride/WARP_SIZE-1)&GetRandomNumber(rng_state);
	for(int safe_counter=0; safe_counter<100000; safe_counter++){
		if (__any(warps_complete.w==(stride/WARP_SIZE)
					||
					kernel_stop_signal!=0) ){
			// все отрешалось, ловить нечего
			break;
		}
		//const int donor_thread=WARP_SIZE*donor_warp+safe+lane;


		rng_state=GetRandomNumber(rng_state^lane);
		// выбираем случайный варп для обмена и от него
		// линейно сканируем направо с wrap'ом
		//if (lane==0 && shard_num/32==63) printf(" \n state %i  donor warp %i", thread_state[shard_num], donor_warp);
		for (int i=0; i<10; i++){
			rng_state=GetRandomNumber(rng_state);
		}
		/*
		const int random_shift=(int)((WARP_SIZE-1)&rng_state);
		const int donor_thread=random_warps_per_lane_base+random_shift;
		*/
		const int donor_thread=WARP_SIZE*
			//((stride/WARP_SIZE-1)&donor_warp)
			//donor_warp
			//((63)&(int)rng_state)
			// база
			lane*(gridDim.x*DEFAULT_BLOCKSIZE/WARP_SIZE/WARP_SIZE)
		+
		//FIXME constants! sizing!
		// рандомный сдвиг 
		(int)((gridDim.x*DEFAULT_BLOCKSIZE/WARP_SIZE-1)&rng_state);
		//lane;
		//if (lane==0) printf(" \n donor thread %i  donor warp %i", donor_thread ,donor_warp);
		assert(donor_thread<(gridDim.x*DEFAULT_BLOCKSIZE));
	
		if(thread_state[shard_num]==ROOT_NOT_READY &&
				TryExchangeWithThread(SDA, donor_thread) 
				){
			//printf("\n t %i try t %i OK", shard_num, donor_thread);
			//atomicAdd(&sd.inspects[donor_thread].w,1);
			CopyRootStateFromThread(SDA, donor_thread);
			__threadfence();
			// разлочиваем тред-донор
			// FIXME  потенциальный баг! при
			// отключении этой строки повторные
			// обмены все равно иногда
			// встречаются!!!!!
			thread_state[donor_thread]=ROOT_NOT_READY;
		}
		__threadfence();
		if (__any(state==FORCED_DECISION)){
			if(lane==0){ atomicAdd(&warps_complete.y,1); }
			if(lane==0){ atomicSub(&warps_complete.w,1); }
			//printf ("\n COPY COMPLETE");
			return true;
		}else{
			if(lane==0){
				atomicAdd(&warps_complete.z,1);
			}
			
			for (int i=0; i<50; i++){
				rng_state=GetRandomNumber(rng_state);
			}
		}
	}
#endif
	return false;
}
__device__ void MainSolverCycle(SDD)
{
	litind conflict_lit = 0;
	int cycles_passed=0;
	int decisions_made=0;
	do{
		//Check SAT
		if(state==BCP_OK && CheckSAT(SDA)){
			state=SAT;
		}
		//Check STOPPED
		if ( inspects.x>=inspects_limit || 
		//if ( inspects.x>=500000|| 
			decisions_made > DECISIONS_LIMIT || 
			kernel_stop_signal!=0){
			state=STOPPED;
		}
		if(__any(state==STOPPED || state==SAT)){  break; }
		if(__all(state==UNSAT)){ break; }

#ifdef FORCE_SERIALIZE
			for(int i=0;i<WARP_SIZE;i++)
				if(i==lane)
#endif
		if (state==BCP_OK){
			MakeNewDecision(SDA);
			decisions_made++;
			state = DEFAULT_DECISION;
		}

#ifdef FORCE_SERIALIZE
			for(int i=0;i<WARP_SIZE;i++)
				if(i==lane)
#endif
		if (state==CONFLICT){
			// Backtracking
			const int shadow_handle = BackjumpFast(SDA, conflict_lit);
			if (shadow_handle==~0){
				state=UNSAT;
			}else{
				//FIXME: оптимизировать и переделать в функцию
				assert(decision_var_index<c.pl_size/2);
				SetVar(SDA, HVarToVar((int)decision_var_index* (1-2*(GetVarphase(SDA, decision_var_index)))), shadow_handle, 1);
				trail[stride*BCP_queue_front+shard_num].var|=SEEN_FLAG;
				decisions_made++;
				state = FORCED_DECISION;
			}
		}
	
#ifdef WORK_SHARE
		ShareWorkInMyWarp(SDA);
#endif
		if(state==DEFAULT_DECISION || state==FORCED_DECISION || state==BCP_STALL){
			//Propagate
#ifdef FORCE_SERIALIZE
			for(int i=0;i<WARP_SIZE;i++)
				if(i==lane)
#endif
			conflict_lit=propagate_thread(SDA, DEFAULT_VOTE_THRESHOLD);
			if (conflict_lit!=0){
				BCPQueueClear(SDA);
				state=CONFLICT;
			}else{
				state = isBCPQueueEmpty(SDA) ?  BCP_OK : BCP_STALL;
			}
		}
	}while(++cycles_passed<=15000000);
}


__global__ void __launch_bounds__(DEFAULT_BLOCKSIZE,8) Solve(
		gpusolverdata sd,
		shadowword* shadow_reasons,
		volatile threadstate* thread_state,
		volatile solver_internal_vars* employer_transfer_vars,
		const int inspects_limit)
{
	int shard_num = threadIdx.x+blockIdx.x*blockDim.x;
#ifdef SHUFFLE_SHARDS
	shard_num = shnums_tmp[shard_num];
#endif
	//int shard_num = threadIdx.x*gridDim.x*gridDim.y+blockIdx.x;
	int4 inspects=sd.inspects[shard_num];
	RESULT state=sd.out[shard_num];
	int trail_end=sd.trail_end[shard_num];
	int decision_var_index=sd.decision_var_index[shard_num];
	int BCP_queue_front=sd.BCP_queue_front[shard_num];
	int stall_count=0;
	int stalled_start, stalled_end;
	varind stalled_var_index;
	var_word wsword;
	varind* vars_heap=sd.vars_heap;


	int shadow_reasons_end = 1;

	const int stride=sd.stride;
	Rcnf& c = sd.c;
	var_word* &ws=sd.ws;
	var_word* &vars=sd.vars;
	trailword* &trail=sd.trail;

	int root_trailpos = trail_end;

	// FIXME аццкий костыль!
	DefineDecisionVar(SDA);
	decision_var_index=0;


	for (int i=0; i<HIDDEN_REASONS_SIZE; i++){ shadow_reasons[shard_num*HIDDEN_REASONS_SIZE+i]=8888; }
	StoreLocalVars(SDA);

	thread_state[shard_num]=ROOT_NOT_READY;
	//if (lane==0)
	do{
		MainSolverCycle(SDA);
		if(lane==0){ 
			atomicAdd(&warps_complete.w,1);
		       	atomicAdd(&warps_complete.x,1);
		}
		if(__any(state==STOPPED || state==SAT) || warps_complete.w==(stride/WARP_SIZE)){
			// все отрешалось, ловить нечего
			thread_state[shard_num]=ROOT_NOT_READY;
			kernel_stop_signal=1;
		}
	}while(kernel_stop_signal==0 && __all(state==UNSAT) &&
			StealWorkFromOtherWarps(SDA));
	// ^ TODO: Сделать так, чтобы от перестановки условий
	// корректность не терялась. 
	if(state!=UNSAT  && state!=SAT){
		state=STOPPED;
	}
	sd.inspects[shard_num].x+=inspects.x;
	sd.inspects[shard_num].y+=inspects.y;
	sd.inspects[shard_num].z+=inspects.z;
	sd.inspects[shard_num].w+=inspects.w;
	sd.out[shard_num]=state;
}


RESULT SolverSolve(
		Rcnf &c,
		Rsolverstate &s,
		var_word* vars,
		var_word* ws,
		trailword* trail,
		const int multi,
		const int inspects_limit ,
		const int kernel_runs_limit ,
		const bool truncate ,
		const int subproblem)
{
	int blocksize =DEFAULT_BLOCKSIZE;
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, 0);

	const int num_inst = (1<<multi);
	// TODO: разобраться с тредами, выходящими за пределы блоков !!!
	//const int num_blocks_truncated = (num_inst - (num_inst%devProp.multiProcessorCount));
	const int num_blocks_truncated = (num_inst - (num_inst%8));
	int num_blocks;
	if (truncate){
		num_blocks = num_blocks_truncated;
	}else{
		num_blocks = num_inst;
	}
	//num_blocks=num_blocks/(blocksize/WARP_SIZE);
	//num_blocks=num_blocks/WARP_SIZE;
	//const int num_blocks = num_blocks_truncated ;

	//int trunc = (blocksize*num_multiprocessors);
	//cout << "\n trunc " << trunc;
	RESULT* out = (RESULT*) malloc(num_inst*sizeof(RESULT));
	
	gpusolverdata sd,sd_temp;
	RESULT result=UNSAT;
	int4* inspects = (int4*) malloc(num_inst*sizeof(int4));
	int4* pp_inspects = (int4*) malloc(num_inst*sizeof(int4));
	double copy_starttime=cpuTime();
	sd.multi=multi;
	sd.stride=1<<multi;
	sd.c.lits_size=c.lits_size;
	sd.c.pl_size=c.pl_size;
	sd.blocks_complete=0;


	//hipMalloc((void**) &qqq, num_inst*sizeof(RESULT));
	//hipMalloc((void**) &sd.out, num_inst*sizeof(RESULT));
	CudaSafeCall(hipMalloc((void**) &sd.out, num_inst*sizeof(RESULT)));
	CudaSafeCall(hipMalloc((void**) &sd.trail_end, num_inst*sizeof(int)));
	CudaSafeCall(hipMalloc((void**) &sd.BCP_queue_front, num_inst*sizeof(int)));
	//CudaSafeCall(hipMalloc((void**) &sd.BCP_queue_back, num_inst*sizeof(int)));
	CudaSafeCall(hipMalloc((void**) &sd.decision_var_index, num_inst*sizeof(int)));
	CudaSafeCall(hipMalloc((void**) &sd.inspects, num_inst*sizeof(int4)));
	//CudaSafeCall(hipMalloc((void**) &sd.conflict_count_old, num_inst*sizeof(int)));
	//CudaSafeCall(hipMalloc((void**) &sd.stall_count, num_inst*sizeof(int)));

	m_CopyToGPU(sd.c.pl, c.pl, c.pl_size*sizeof(litind));
	m_CopyToGPU(sd.c.lits, c.lits, c.lits_size*sizeof(uintx));

	CudaSafeCall(hipMalloc((void**) &sd.ws, num_inst*sizeof(var_word)*s.ws_size));
	CudaSafeCall(hipMalloc((void**) &sd.vars, num_inst*sizeof(var_word)*s.vars_size));
	CudaSafeCall(hipMalloc((void**) &sd.trail, num_inst*sizeof(trailword)*c.pl_size/2));
	CudaSafeCall(hipMalloc((void**) &sd.vars_heap,	num_inst*sizeof(varind)*c.pl_size/2));

	shadowword* shadow_reasons = NULL;
	CudaSafeCall(hipMalloc((void**) &shadow_reasons, HIDDEN_REASONS_SIZE*num_inst*sizeof(shadowword)));

	volatile solver_internal_vars* employer_transfer_vars;
	CudaSafeCall(hipMalloc((void**)&employer_transfer_vars, num_inst*sizeof(solver_internal_vars)));

	volatile threadstate* thread_state;
	CudaSafeCall(hipMalloc((void**)&thread_state, (num_inst)*sizeof(threadstate)));

	CudaSafeCall(hipMalloc((void**) &sd.trail, num_inst*sizeof(trailword)*c.pl_size/2));

	int* partial_warps = NULL;
	//CudaSafeCall(hipMalloc((void**) &partial_warps, WARP_SIZE*sizeof(int)));
	//CudaSafeCall(hipMalloc((void**) &sd.bcpstat, num_inst*sizeof(int)*BCPSTATSIZE));

	sd_temp=sd;
	//докопируем начальное заполнение

	m_CopyToGPU(sd_temp.ws, ws, sizeof(var_word)*s.ws_size);
	m_CopyToGPU(sd_temp.vars, vars, sizeof(var_word)*s.vars_size);
	m_CopyToGPU(sd_temp.trail, trail, sizeof(trailword)*c.pl_size/2);

	CudaSafeCall(hipBindTexture(0, texPl, sd.c.pl, c.pl_size*sizeof(litind)));

	hipEvent_t kernel_start, kernel_stop;
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);
	hipEventRecord(kernel_start);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(PreprocessR), CACHE_MODE );
	PreprocessR <<< num_blocks/DEFAULT_BLOCKSIZE, blocksize >>>
		(sd, shadow_reasons, thread_state, sd_temp, s.trail_end,
		 s.BCP_queue_front, partial_warps,
		 employer_transfer_vars, subproblem);
	//PreprocessR <<< 1, 32 >>> (sd,sd_temp, s.trail_end);

	hipDeviceSynchronize ();
	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);
	float preprocessTime;
	hipEventElapsedTime(&preprocessTime, kernel_start, kernel_stop);

	CudaCheckError();
	CudaSafeCall(hipMemcpy((void*) pp_inspects, sd.inspects, num_inst*sizeof(int4), hipMemcpyDeviceToHost));
	CudaSafeCall(hipFree(sd_temp.ws));
	CudaSafeCall(hipFree(sd_temp.vars));
	CudaSafeCall(hipFree(sd_temp.trail));

	//int int_zero=0;
	//m_CopyToGPU(sd.blocks_complete, &int_zero, sizeof(int));
	
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);
	hipEventRecord(kernel_start);



	int kernel_runs;
	
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(Solve), CACHE_MODE);
	for (kernel_runs=0; kernel_runs<kernel_runs_limit; kernel_runs++){
		hipDeviceSynchronize ();
		//printf("\n %3i %4i %6i %6i %6.0f", blocksize, num_blocks_truncated, kernel_runs, inspects_limit);

		int (*bcpstat)[BCPSTATSIZE] = (int(*)[BCPSTATSIZE]) malloc(num_inst*sizeof(int)*BCPSTATSIZE);
		int4 h_shards_complete={0,0,0,0};

		CudaSafeCall(hipStreamCreate(&solver_kernel_stream));
		CudaSafeCall(hipStreamCreate(&signal_stream));
		Solve <<< num_blocks/DEFAULT_BLOCKSIZE, blocksize, 0, solver_kernel_stream>>> (sd, shadow_reasons, thread_state, employer_transfer_vars, inspects_limit);
		signal(SIGINT, INTHandler);
		int h_stop_flag=0;
		while(1){
			hipMemcpyFromSymbolAsync(&h_stop_flag, HIP_SYMBOL(kernel_stop_signal),
						sizeof(int), 0,
						hipMemcpyDeviceToHost,
						signal_stream);
			if (h_stop_flag==1){
				break;
			}
			if (e_flag==1){
				const int stop_mark=1;
				hipMemcpyToSymbolAsync(HIP_SYMBOL(kernel_stop_signal), (void*)
							&stop_mark,
							sizeof(int), 0,
							hipMemcpyHostToDevice, signal_stream);
				break;
			}
			SleepSomeTime(50000000);
		}
		CudaCheckError();

		CudaSafeCall(hipStreamDestroy(solver_kernel_stream));
		CudaSafeCall(hipStreamDestroy(signal_stream));

		CudaSafeCall(hipMemcpyFromSymbol(&h_shards_complete, HIP_SYMBOL(shards_complete),
					sizeof(int4), 0,
					hipMemcpyDeviceToHost));
		
		printf("\n Cycles %i  Complete shards: %i %i %i",
				h_shards_complete.w,
				h_shards_complete.x, h_shards_complete.y, h_shards_complete.z);
		

		int4 h_warps_complete={0,0,0,0};
		CudaSafeCall(hipMemcpyFromSymbol(&h_warps_complete, HIP_SYMBOL(warps_complete),
					sizeof(int4), 0,
					hipMemcpyDeviceToHost));
		printf("\n Warps [comlete, stealings, failed st., total] : %i %i %i %i",
				h_warps_complete.x,
				h_warps_complete.y,
				h_warps_complete.z,
				h_warps_complete.w
				);
		/*
		printf("\n");
		for (int i=0; i<num_inst; i++){
			for (int j=0; j<=bcpstat[i][0]; j++){
				printf("%i ", bcpstat[i][j]);
			}
			printf("\n");
		}
		*/

		//SolveR <<<1, 1>>> (sd, inspects_limit);
		//SolveR <<<1,blocksize>>> (sd, inspects_limit);
		CudaSafeCall(hipMemcpy((void*) out, sd.out, num_inst*sizeof(RESULT), hipMemcpyDeviceToHost));

		RESULT batch_state =UNSAT;
		for (int i=0;i<num_inst;i++){
			switch (out[i]){
				case SAT:
					result = SAT;batch_state = SAT;
					break;
				case STOPPED:
					result = STOPPED; batch_state = STOPPED;
				       	//break;
			}
		}
		if (batch_state == UNSAT || batch_state == SAT){
			break;
		}
		free(bcpstat);
	}
	
	hipDeviceSynchronize ();
	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);
	float solverTime;
	hipEventElapsedTime(&solverTime, kernel_start, kernel_stop);

	//int*	partial_warps_host=(int*) malloc(WARP_SIZE*sizeof(int));
	//CudaSafeCall(hipMemcpy((void*) partial_warps_host, partial_warps, WARP_SIZE*sizeof(int), hipMemcpyDeviceToHost));


	
	/*
	printf("\n Warp sizes counts 0-31:");
	for(int i=0;i<WARP_SIZE;i++)
		printf(" %7i",partial_warps_host[i]);
	printf("\n");
	for(int i=2;i<WARP_SIZE;i++)
		partial_warps_host[i]+=partial_warps_host[i-1];
	int partial_warps_total=partial_warps_host[WARP_SIZE-1];
	int warps_total=partial_warps_host[0];
	printf("\n %i\/%i (%i%%) partial warps",partial_warps_total, warps_total, (partial_warps_total*100)/warps_total);
		
	free(partial_warps_host);
	*/

//	printf("\n Blocksize Blocks  Kernel_runs Inspects_limit  Total_runtime(ms)");
	
	CudaSafeCall(hipMemcpy((void*) out, sd.out, num_inst*sizeof(RESULT), hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy((void*) inspects, sd.inspects, num_inst*sizeof(int4), hipMemcpyDeviceToHost));

	var_word* output_vars = (var_word*) malloc(num_inst*sizeof(var_word)*s.vars_size);
	CudaSafeCall(hipMemcpy((void*) output_vars, sd.vars, num_inst*sizeof(var_word)*s.vars_size, hipMemcpyDeviceToHost));

	
/*
	for(int i=0;i<num_inst;i++)
		cout << "\n" << inspects[i];
*/
	CudaSafeCall(hipFree(sd.out));
	CudaSafeCall(hipFree(sd.trail_end));
	CudaSafeCall(hipFree(sd.BCP_queue_front));
	//CudaSafeCall(hipFree(sd.BCP_queue_back));
	//CudaSafeCall(hipFree(sd.BCP_queue));
	CudaSafeCall(hipFree(sd.decision_var_index));
	CudaSafeCall(hipFree(sd.inspects));
	CudaSafeCall(hipFree(sd.ws));
	CudaSafeCall(hipFree(sd.trail));
	CudaSafeCall(hipFree(sd.vars));
	//CudaSafeCall(hipFree(sd.conflict_count_old));

	//CudaSafeCall(hipFree(sd.bcpstat));

	CudaSafeCall(hipFree(sd.c.pl));
	CudaSafeCall(hipFree(sd.c.lits));

	//CudaSafeCall(hipFree(sd.blocks_complete));

	//CudaSafeCall(hipFree(partial_warps));

	CudaSafeCall(hipFree((void*)thread_state));
	CudaSafeCall(hipFree((void*)employer_transfer_vars));
	int sat=0;
	int stopped=0;
	int unsat=0;
	int satthread=0;
	for(int i=0;i<num_inst;i++){
		if(out[i]==SAT){
			satthread=i;
			sat++;
			for (int m=0; m<c.pl_size/WORD_SIZE+1; m++){
				vars[m]=output_vars[num_inst*m+i];
			}
			
		}else if (out[i]==UNSAT){
			unsat++;
		}else if (out[i]==STOPPED){
			stopped++;
		}
	}
	free(output_vars);

	if(sat>0){
		result=SAT;
	}else if (stopped>0){
		result=STOPPED;
	}else if (unsat==num_inst){
		result=UNSAT;
	}else{
		result=ERROR;
	}

	long long int total_inspects=0;
	long long int pp_total_inspects=0;
	for(int i=0;i<num_inst;i++){
		//if(out[i]==SAT || out[i]==UNSAT){
		{
			printf("\n %6i %7i %7i %7f sh %7i ext %7i", i, inspects[i].x, inspects[i].y,
					100*(float)inspects[i].y/(float)inspects[i].x,
					inspects[i].z, inspects[i].w-inspects[i].z);
		}
		pp_total_inspects+=pp_inspects[i].x;
		total_inspects+=inspects[i].x;
		    if(out[i]==SAT){printf(" S");}
		    if(out[i]==UNSAT){printf(" U");}

	}

	
	printf("\n PP    Stats: %6.2f MIns/sec ",(total_inspects-pp_total_inspects)/1000000/preprocessTime*1000);
	printf("\n Solve Stats: %6.2f MIns/sec ",(total_inspects)/1000000/solverTime*1000);
	printf("\n Results: %3i %5i %6i %6i %9i %6.0f %6.0f  %4i %4i %4i",blocksize, num_blocks,
			kernel_runs, inspects_limit, total_inspects, preprocessTime,
			solverTime, sat, unsat, stopped);
	if (sat>0){
		printf("  sat thread number %5i inspects %i", satthread,
				inspects[satthread].x);
	}
	
/*
	for(int i=0;i<num_inst;i++){
		if(out[i]==SAT){
			cout << "1";
			result = SAT;
		}else{
			if (out[i]==UNSAT)
				cout << "0";
			else
				if (out[i]==STOPPED)
					cout << "P";
				else
					cout << "E";
		}
	}
*/	
	free(out);
	free(inspects);
	free(pp_inspects);
	return result;
}
